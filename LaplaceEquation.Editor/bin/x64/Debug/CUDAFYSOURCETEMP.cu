
#include <hip/hip_runtime.h>

// MyCudafy.CudafyMultiDimentionalArray
extern "C" __global__ void LaplaceSolver( double* prev, int prevLen0,  double* next, int nextLen0,  int* sizes, int sizesLen0,  int* extV, int extVLen0,  int* intV, int intVLen0,  double* w, int wLen0);
// MyCudafy.CudafyMultiDimentionalArray
extern "C" __global__ void Copy( double* prev, int prevLen0,  double* next, int nextLen0);
// MyCudafy.CudafyMultiDimentionalArray
extern "C" __global__ void Delta( double* prev, int prevLen0,  double* next, int nextLen0,  double* delta, int deltaLen0);
// MyCudafy.CudafyMultiDimentionalArray
extern "C" __global__ void Max( double* prev, int prevLen0,  double* next, int nextLen0);

// MyCudafy.CudafyMultiDimentionalArray
__constant__ double _a[100];
#define _aLen0 100
// MyCudafy.CudafyMultiDimentionalArray
__constant__ double _b[1];
#define _bLen0 1
// MyCudafy.CudafyMultiDimentionalArray
__constant__ int _sizes[2];
#define _sizesLen0 2
// MyCudafy.CudafyMultiDimentionalArray
__constant__ double _lengths[2];
#define _lengthsLen0 2
// MyCudafy.CudafyMultiDimentionalArray
__constant__ int _intV[3];
#define _intVLen0 3
// MyCudafy.CudafyMultiDimentionalArray
__constant__ int _extV[3];
#define _extVLen0 3
// MyCudafy.CudafyMultiDimentionalArray
__constant__ double _w[3];
#define _wLen0 3
// MyCudafy.CudafyMultiDimentionalArray
extern "C" __global__ void LaplaceSolver( double* prev, int prevLen0,  double* next, int nextLen0,  int* sizes, int sizesLen0,  int* extV, int extVLen0,  int* intV, int intVLen0,  double* w, int wLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < intV[(sizesLen0)]; i += blockDim.x * gridDim.x)
	{
		int num = 0;
		int j = 0;
		int num2 = i;
		while (j < sizesLen0)
		{
			num += (1 + num2 % (sizes[(j)] - 2)) * extV[(j)];
			num2 /= sizes[(j)] - 2;
			j++;
		}
		double num3 = 0.0;
		for (j = 0; j < sizesLen0; j++)
		{
			num3 += (prev[(num - extV[(j)])] + prev[(num + extV[(j)])]) * w[(j)];
		}
		next[(num)] = num3 - prev[(num)] * w[(sizesLen0)];
	}
}
// MyCudafy.CudafyMultiDimentionalArray
extern "C" __global__ void Copy( double* prev, int prevLen0,  double* next, int nextLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < prevLen0; i += blockDim.x * gridDim.x)
	{
		next[(i)] = prev[(i)];
	}
}
// MyCudafy.CudafyMultiDimentionalArray
extern "C" __global__ void Delta( double* prev, int prevLen0,  double* next, int nextLen0,  double* delta, int deltaLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < prevLen0; i += blockDim.x * gridDim.x)
	{
		delta[(i)] = ((next[(i)] > prev[(i)]) ? (next[(i)] - prev[(i)]) : (prev[(i)] - next[(i)]));
	}
}
// MyCudafy.CudafyMultiDimentionalArray
extern "C" __global__ void Max( double* prev, int prevLen0,  double* next, int nextLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nextLen0; i += blockDim.x * gridDim.x)
	{
		next[(i)] = 0.0;
		int num = (prevLen0 + nextLen0 - 1) / nextLen0;
		while (num-- > 0)
		{
			int num2 = num * nextLen0 + i;
			if (num2 < prevLen0 && prev[(num2)] > next[(i)])
			{
				next[(i)] = prev[(num2)];
			}
		}
	}
}
