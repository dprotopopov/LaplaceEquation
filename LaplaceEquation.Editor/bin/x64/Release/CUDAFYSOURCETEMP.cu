
#include <hip/hip_runtime.h>

// MyCudafy.CudafyMulti
extern "C" __global__ void LaplaceSolver( double* prev, int prevLen0,  double* next, int nextLen0,  int* sizes, int sizesLen0,  int* extV, int extVLen0,  int* intV, int intVLen0,  double* w, int wLen0);
// MyCudafy.CudafyMulti
extern "C" __global__ void Copy( double* prev, int prevLen0,  double* next, int nextLen0);
// MyCudafy.CudafyMulti
extern "C" __global__ void Square( double* prev, int prevLen0,  double* next, int nextLen0,  double* delta, int deltaLen0);
// MyCudafy.CudafyMulti
extern "C" __global__ void Delta( double* prev, int prevLen0,  double* next, int nextLen0,  double* delta, int deltaLen0);
// MyCudafy.CudafyMulti
extern "C" __global__ void Max( double* prev, int prevLen0,  double* next, int nextLen0);
// MyCudafy.CudafyMulti
extern "C" __global__ void Sum( double* prev, int prevLen0,  double* next, int nextLen0);

// MyCudafy.CudafyMulti
__constant__ double _a[100];
#define _aLen0 100
// MyCudafy.CudafyMulti
__constant__ double _b[1];
#define _bLen0 1
// MyCudafy.CudafyMulti
__constant__ int _sizes[2];
#define _sizesLen0 2
// MyCudafy.CudafyMulti
__constant__ double _lengths[2];
#define _lengthsLen0 2
// MyCudafy.CudafyMulti
__constant__ int _intV[3];
#define _intVLen0 3
// MyCudafy.CudafyMulti
__constant__ int _extV[3];
#define _extVLen0 3
// MyCudafy.CudafyMulti
__constant__ double _w[3];
#define _wLen0 3
// MyCudafy.CudafyMulti
extern "C" __global__ void LaplaceSolver( double* prev, int prevLen0,  double* next, int nextLen0,  int* sizes, int sizesLen0,  int* extV, int extVLen0,  int* intV, int intVLen0,  double* w, int wLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < intV[(sizesLen0)]; i += blockDim.x * gridDim.x)
	{
		int num = 0;
		int j = 0;
		int num2 = i;
		while (j < sizesLen0)
		{
			num += (1 + num2 % (sizes[(j)] - 2)) * extV[(j)];
			num2 /= sizes[(j)] - 2;
			j++;
		}
		double num3 = prev[(num)] * w[(sizesLen0)];
		for (int k = 0; k < sizesLen0; k++)
		{
			num3 += (prev[(num - extV[(k)])] + prev[(num + extV[(k)])]) * w[(k)];
		}
		next[(num)] = num3;
	}
}
// MyCudafy.CudafyMulti
extern "C" __global__ void Copy( double* prev, int prevLen0,  double* next, int nextLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < prevLen0; i += blockDim.x * gridDim.x)
	{
		next[(i)] = prev[(i)];
	}
}
// MyCudafy.CudafyMulti
extern "C" __global__ void Square( double* prev, int prevLen0,  double* next, int nextLen0,  double* delta, int deltaLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < prevLen0; i += blockDim.x * gridDim.x)
	{
		double num = next[(i)];
		num *= num;
		delta[(i)] = num;
	}
}
// MyCudafy.CudafyMulti
extern "C" __global__ void Delta( double* prev, int prevLen0,  double* next, int nextLen0,  double* delta, int deltaLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < prevLen0; i += blockDim.x * gridDim.x)
	{
		double num = next[(i)] * (prev[(i)] - next[(i)]);
		num *= num;
		delta[(i)] = num;
	}
}
// MyCudafy.CudafyMulti
extern "C" __global__ void Max( double* prev, int prevLen0,  double* next, int nextLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nextLen0; i += blockDim.x * gridDim.x)
	{
		next[(i)] = 0.0;
		int num = 0;
		while (num * nextLen0 + i < prevLen0)
		{
			int num2 = num * nextLen0 + i;
			if (prev[(num2)] > next[(i)])
			{
				next[(i)] = prev[(num2)];
			}
			num++;
		}
	}
}
// MyCudafy.CudafyMulti
extern "C" __global__ void Sum( double* prev, int prevLen0,  double* next, int nextLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nextLen0; i += blockDim.x * gridDim.x)
	{
		next[(i)] = 0.0;
		int num = 0;
		while (num * nextLen0 + i < prevLen0)
		{
			int num2 = num * nextLen0 + i;
			next[(i)] += prev[(num2)];
			num++;
		}
	}
}
