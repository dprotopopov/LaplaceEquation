
#include <hip/hip_runtime.h>

// MyCudafy.CudafyMulti
extern "C" __global__ void LaplaceSolver( double* prev, int prevLen0,  double* next, int nextLen0,  int* sizes, int sizesLen0,  int* extV, int extVLen0,  int* intV, int intVLen0,  double* w, int wLen0,  double* b, int bLen0,  double* c, int cLen0);
// MyCudafy.CudafyMulti
extern "C" __global__ void LaplaceSolverWithRelax( double* array, int arrayLen0,  int* sizes, int sizesLen0,  int* extV, int extVLen0,  int* intV, int intVLen0,  double* w, int wLen0,  double* b, int bLen0,  double* c, int cLen0, int p);
// MyCudafy.CudafyMulti
extern "C" __global__ void Copy( double* prev, int prevLen0,  double* next, int nextLen0);
// MyCudafy.CudafyMulti
extern "C" __global__ void Clear( double* array, int arrayLen0);
// MyCudafy.CudafyMulti
extern "C" __global__ void Square( double* prev, int prevLen0,  double* next, int nextLen0,  double* delta, int deltaLen0);
// MyCudafy.CudafyMulti
extern "C" __global__ void Delta( double* prev, int prevLen0,  double* next, int nextLen0,  double* delta, int deltaLen0);
// MyCudafy.CudafyMulti
extern "C" __global__ void Max( double* prev, int prevLen0,  double* next, int nextLen0);
// MyCudafy.CudafyMulti
extern "C" __global__ void Sum( double* prev, int prevLen0,  double* next, int nextLen0);

// MyCudafy.CudafyMulti
__constant__ double _a[100];
#define _aLen0 100
// MyCudafy.CudafyMulti
__constant__ double _b[1];
#define _bLen0 1
// MyCudafy.CudafyMulti
__constant__ double _c[1];
#define _cLen0 1
// MyCudafy.CudafyMulti
__constant__ int _sizes[2];
#define _sizesLen0 2
// MyCudafy.CudafyMulti
__constant__ double _lengths[2];
#define _lengthsLen0 2
// MyCudafy.CudafyMulti
__constant__ int _intV[3];
#define _intVLen0 3
// MyCudafy.CudafyMulti
__constant__ int _extV[3];
#define _extVLen0 3
// MyCudafy.CudafyMulti
__constant__ double _w[3];
#define _wLen0 3
// MyCudafy.CudafyMulti
extern "C" __global__ void LaplaceSolver( double* prev, int prevLen0,  double* next, int nextLen0,  int* sizes, int sizesLen0,  int* extV, int extVLen0,  int* intV, int intVLen0,  double* w, int wLen0,  double* b, int bLen0,  double* c, int cLen0)
{
	double num = 0.0;
	double num2 = 0.0;
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < intV[(sizesLen0)]; i += blockDim.x * gridDim.x)
	{
		int num3 = 0;
		int j = 0;
		int num4 = i;
		while (j < sizesLen0)
		{
			int num5 = 1 + num4 % (sizes[(j)] - 2);
			num3 += num5 * extV[(j)];
			num4 /= sizes[(j)] - 2;
			j++;
		}
		double num6 = prev[(num3)];
		double num7 = num6 * w[(sizesLen0)];
		for (int k = 0; k < sizesLen0; k++)
		{
			num7 += (prev[(num3 - extV[(k)])] + prev[(num3 + extV[(k)])]) * w[(k)];
		}
		next[(num3)] = num7;
		double num8 = num6 - num7;
		double num9 = num6 + num7;
		num8 *= num8;
		num9 *= num9;
		num += num8;
		num2 += num9;
	}
	b[(blockDim.x * blockIdx.x + threadIdx.x)] = num;
	c[(blockDim.x * blockIdx.x + threadIdx.x)] = num2;
}
// MyCudafy.CudafyMulti
extern "C" __global__ void LaplaceSolverWithRelax( double* array, int arrayLen0,  int* sizes, int sizesLen0,  int* extV, int extVLen0,  int* intV, int intVLen0,  double* w, int wLen0,  double* b, int bLen0,  double* c, int cLen0, int p)
{
	double num = b[(blockDim.x * blockIdx.x + threadIdx.x)];
	double num2 = c[(blockDim.x * blockIdx.x + threadIdx.x)];
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < intV[(sizesLen0)]; i += blockDim.x * gridDim.x)
	{
		int num3 = 0;
		int num4 = 0;
		int j = 0;
		int num5 = i;
		while (j < sizesLen0)
		{
			int num6 = 1 + num5 % (sizes[(j)] - 2);
			num4 += num6;
			num3 += num6 * extV[(j)];
			num5 /= sizes[(j)] - 2;
			j++;
		}
		if (num4 % 2 == p)
		{
			double num7 = array[(num3)];
			double num8 = num7 * w[(sizesLen0)];
			for (int k = 0; k < sizesLen0; k++)
			{
				num8 += (array[(num3 - extV[(k)])] + array[(num3 + extV[(k)])]) * w[(k)];
			}
			array[(num3)] = num8;
			double num9 = num7 - num8;
			double num10 = num7 + num8;
			num9 *= num9;
			num10 *= num10;
			num += num9;
			num2 += num10;
		}
	}
	b[(blockDim.x * blockIdx.x + threadIdx.x)] = num;
	c[(blockDim.x * blockIdx.x + threadIdx.x)] = num2;
}
// MyCudafy.CudafyMulti
extern "C" __global__ void Copy( double* prev, int prevLen0,  double* next, int nextLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < prevLen0; i += blockDim.x * gridDim.x)
	{
		next[(i)] = prev[(i)];
	}
}
// MyCudafy.CudafyMulti
extern "C" __global__ void Clear( double* array, int arrayLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < arrayLen0; i += blockDim.x * gridDim.x)
	{
		array[(i)] = 0.0;
	}
}
// MyCudafy.CudafyMulti
extern "C" __global__ void Square( double* prev, int prevLen0,  double* next, int nextLen0,  double* delta, int deltaLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < prevLen0; i += blockDim.x * gridDim.x)
	{
		double num = next[(i)];
		num *= num;
		delta[(i)] = num;
	}
}
// MyCudafy.CudafyMulti
extern "C" __global__ void Delta( double* prev, int prevLen0,  double* next, int nextLen0,  double* delta, int deltaLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < prevLen0; i += blockDim.x * gridDim.x)
	{
		double num = next[(i)] * (prev[(i)] - next[(i)]);
		num *= num;
		delta[(i)] = num;
	}
}
// MyCudafy.CudafyMulti
extern "C" __global__ void Max( double* prev, int prevLen0,  double* next, int nextLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nextLen0; i += blockDim.x * gridDim.x)
	{
		next[(i)] = 0.0;
		int num = 0;
		while (num * nextLen0 + i < prevLen0)
		{
			int num2 = num * nextLen0 + i;
			if (prev[(num2)] > next[(i)])
			{
				next[(i)] = prev[(num2)];
			}
			num++;
		}
	}
}
// MyCudafy.CudafyMulti
extern "C" __global__ void Sum( double* prev, int prevLen0,  double* next, int nextLen0)
{
	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < nextLen0; i += blockDim.x * gridDim.x)
	{
		next[(i)] = 0.0;
		int num = 0;
		while (num * nextLen0 + i < prevLen0)
		{
			int num2 = num * nextLen0 + i;
			next[(i)] += prev[(num2)];
			num++;
		}
	}
}
